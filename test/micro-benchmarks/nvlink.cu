#include "hip/hip_runtime.h"
#include <groute/communication.h>
#include <groute/event_pool.h>
#include <groute/link.h>
#include <groute/policy.h>
#include <groute/router.h>
#include <groute/worklist.h>
#include <thrust/device_vector.h>
#include <utils/cuda_utils.h>
#include <utils/stopwatch.h>

#include <cub/grid/grid_barrier.cuh>
#include <sstream>

DEFINE_int32(data_size, 1024, "Start with a specific number of GPUs");
DEFINE_int32(max_number, 10, "Start with a specific number of GPUs");
DEFINE_int32(chunk_size, 1024, "Start with a specific number of GPUs");
DEFINE_string(dst_dev, "0", "");
DEFINE_int32(nrings, 1, "number of rings");

__global__ static void copyp2p(void* dest, const void* src, size_t size) {
  size_t globalId = blockIdx.x * blockDim.x + threadIdx.x;
  size_t gridSize = blockDim.x * gridDim.x;
  auto unit_size = sizeof(int4);
  auto num_elems = size / unit_size;
  auto rest_bytes = size % unit_size;

  auto* dest_4 = (int4*) dest;
  auto* src_4 = (const int4*) src;

#pragma unroll
  for (size_t i = globalId; i < num_elems; i += gridSize) {
    dest_4[i] = src_4[i];
  }

  auto* dest_c = (char*) dest + size - rest_bytes;
  auto* src_c = (const char*) src + size - rest_bytes;

#pragma unroll
  for (size_t i = globalId; i < rest_bytes; i += gridSize) {
    dest_c[i] = src_c[i];
  }
}

#include <condition_variable>
#include <deque>
#include <mutex>

template <typename T>
class BlockingQueue {
 private:
  std::mutex d_mutex;
  std::condition_variable d_condition;
  std::deque<T> d_queue;

 public:
  void push(T const& value) {
    {
      std::unique_lock<std::mutex> lock(this->d_mutex);
      d_queue.push_front(value);
    }
    this->d_condition.notify_one();
  }

  T pop() {
    std::unique_lock<std::mutex> lock(this->d_mutex);
    this->d_condition.wait(lock, [=] { return !this->d_queue.empty(); });
    T rc(std::move(this->d_queue.back()));
    this->d_queue.pop_back();
    return rc;
  }
};

template <typename T>
void FilterAndSplitSegment(
    groute::Stream& stream, groute::Segment<T>& segment, int limit,
    std::vector<std::shared_ptr<groute::Worklist<T>>>& wl) {
  size_t seg_size = segment.GetSegmentSize();
  T* data = segment.GetSegmentPtr();
  size_t num_split = wl.size();

  for (int idx = 0; idx < num_split; idx++) {
    groute::dev::Worklist<T> d_dev_wl = wl[idx]->DeviceObject();

    LaunchKernel(stream, seg_size, [=] __device__() {
      auto tid = TID_1D;
      auto nthreads = TOTAL_THREADS_1D;

      for (int i = 0 + tid; i < seg_size; i += nthreads) {
        auto key = data[i].first;
        auto val = data[i].second;

        if (val < limit) {
          if (idx == key % num_split) {
            d_dev_wl.append_warp(thrust::make_pair(key, val + 1));
          }
        }
      }
    });
  }
}

void Rings() {
  int root = 0;
  int ngpus;
  int data_size = FLAGS_data_size;
  int chunk_size = FLAGS_chunk_size;
  int num_buffers = 5;
  int max_number = FLAGS_max_number;
  using data_type = thrust::pair<int, int>;

  GROUTE_CUDA_CHECK(hipGetDeviceCount(&ngpus));

  groute::Context context(ngpus);
  auto policy = groute::router::Policy::CreateMultiRingsPolicy(ngpus);
  groute::router::Router<data_type> router(context, policy);
  std::vector<groute::Link<data_type>> out_links;
  std::vector<groute::Link<data_type>> in_links;
  std::vector<std::shared_ptr<BlockingQueue<groute::Segment<data_type>>>>
      segs_for_send;
  std::vector<std::vector<std::shared_ptr<groute::Worklist<data_type>>>> tmp_wl;

  tmp_wl.resize(ngpus);

  int nring = policy->GetRouteNum();
  std::cout << "Ring num: " << nring << std::endl;

  // Init
  for (int dev = 0; dev < ngpus; dev++) {
    GROUTE_CUDA_CHECK(hipSetDevice(dev));
    in_links.emplace_back(router, dev, chunk_size, num_buffers);
    out_links.emplace_back(dev, router);
    segs_for_send.push_back(
        std::make_shared<BlockingQueue<groute::Segment<data_type>>>());
    for (int ring = 0; ring < nring; ring++) {
      tmp_wl[dev].push_back(
          std::make_shared<groute::Worklist<data_type>>(data_size));
    }
  }

  thrust::host_vector<data_type> host_buffer(data_size);

  for (int i = 0; i < host_buffer.size(); i++) {
    host_buffer[i] = thrust::make_pair(i, 0);
  }

  thrust::device_vector<data_type> dev_buffer(host_buffer.begin(),
                                              host_buffer.end());

  groute::Segment<data_type> seg(thrust::raw_pointer_cast(dev_buffer.data()),
                                 data_size);
  seg.metadata = 0;
  out_links[root].Send(seg, groute::Event());

  bool running = true;

  std::vector<std::thread> threads;
  std::atomic<int32_t> remain_elems(data_size);
  Stopwatch sw;

  sw.start();

  for (int dev = 0; dev < ngpus; dev++) {
    threads.emplace_back(
        [&](int curr_dev) {
          GROUTE_CUDA_CHECK(hipSetDevice(curr_dev));
          groute::Stream stream(curr_dev);
          auto& in_link = in_links[curr_dev];
          auto& worklists = tmp_wl[curr_dev];

          while (running) {
            auto fut = in_link.Receive();
            auto seg = fut.get();
            // seg will be empty when router's shutdown method is invoked
            if (seg.Empty())
              break;

            // waiting for receive are done
            seg.Wait(stream.hip_stream);

            for (auto& wl : worklists) {
              wl->ResetAsync(stream.hip_stream);
            }

            int finished_size = seg.GetSegmentSize();

            FilterAndSplitSegment(stream, seg, max_number, worklists);

            groute::Event split_ev =
                context.RecordEvent(curr_dev, stream.hip_stream);

            in_link.ReleaseBuffer(seg, split_ev);

            std::stringstream ss;
            ss << "Dev: " << curr_dev << " Seg size: ";
            for (int ring = 0; ring < worklists.size(); ring++) {
              auto split_seg = worklists[ring]->ToSeg(stream);
              ss << split_seg.GetSegmentSize() << " ";
              if (!split_seg.Empty()) {
                finished_size -= split_seg.GetSegmentSize();
                split_seg.metadata = 0;
                segs_for_send[curr_dev]->push(split_seg);
              }
            }
            ss << std::endl;

            std::cout << ss.str();

            int n_remain =
                remain_elems.fetch_sub(finished_size) - finished_size;

            if (n_remain <= 0) {
              running = false;
              std::cout << "Done!" << std::endl;
              // notify all senders to exit
              for (int i_dev = 0; i_dev < ngpus; i_dev++) {
                segs_for_send[i_dev]->push(groute::Segment<data_type>());
              }
              router.Shutdown();
            }
          }
        },
        dev);

    threads.emplace_back(
        [&](int curr_dev) {
          GROUTE_CUDA_CHECK(hipSetDevice(curr_dev));
          groute::Stream stream(curr_dev);
          auto& out_link = out_links[curr_dev];

          while (running) {
            auto seg = segs_for_send[curr_dev]->pop();
            if (seg.Empty()) {
              std::cout << "Sender " << curr_dev << " exit." << std::endl;
              break;
            }
            auto sent_ev = out_link.Send(seg, groute::Event()).get();
            sent_ev.Sync();
          }
        },
        dev);
  }

  for (auto& th : threads) {
    th.join();
  }
  sw.stop();

  std::cout << "Running time: " << sw.ms() << std::endl;
}

void TestSR(std::vector<int>& dsts) {
  using data_type = int32_t;
  size_t size = 1024 * 1024 * 1024;
  size_t chunk_size = 8 * 1024 * 1024;
  size_t num_buffer = 10;
  float size_in_mb = size * sizeof(data_type) / 1024.0f / 1024;
  int src = 0;

  int ngpus;
  GROUTE_CUDA_CHECK(hipGetDeviceCount(&ngpus));

  groute::Context context(ngpus);
  auto policy = groute::router::Policy::CreateScatterPolicy(src, dsts);
  groute::router::Router<data_type> router(context, policy);
  auto sender = router.GetSender(src);
  //  auto receiver = router.GetReceiver(dst);

  context.SetDevice(src);
  thrust::device_vector<data_type> send_data(size);
  std::map<int, std::unique_ptr<thrust::device_vector<data_type>>>
      recv_data_vec;
  std::map<int, std::unique_ptr<groute::router::IPipelinedReceiver<data_type>>>
      receivers;
  for (auto dst : dsts) {
    context.SetDevice(dst);
    recv_data_vec[dst] =
        std::make_unique<thrust::device_vector<data_type>>(size);
    receivers[dst] =
        router.CreatePipelinedReceiver(dst, chunk_size, num_buffer);
  }
  std::vector<std::thread> threads;
  Stopwatch sw;

  for (auto dst : dsts) {
    GROUTE_CUDA_CHECK(hipSetDevice(dst));
    groute::Stream stream(dst);
    sw.start();
    GROUTE_CUDA_CHECK(hipMemcpyAsync(
        thrust::raw_pointer_cast(recv_data_vec[dst]->data()),
        thrust::raw_pointer_cast(send_data.data()), sizeof(data_type) * size,
        hipMemcpyDeviceToDevice, stream.hip_stream));
    stream.Sync();
    sw.stop();
    std::cout << src << "->" << dst << " Size: " << size_in_mb << " MB"
              << " Copy time: " << sw.ms()
              << " Bandwidth: " << size_in_mb / (sw.ms() / 1000) << " MB/s"
              << std::endl;
  }

  sw.start();

  threads.emplace_back([&]() {
    context.SetDevice(src);
    groute::Segment<data_type> seg(thrust::raw_pointer_cast(send_data.data()),
                                   send_data.size());
    auto send_ready = sender->Send(seg, groute::Event()).get();
    send_ready.Sync();
    std::cout << "Shutdown router" << std::endl;
    router.Shutdown();
  });

  std::atomic<int> total_recv_size(0);

  for (auto dst : dsts) {
    threads.emplace_back([&, dst]() {
      context.SetDevice(dst);
      auto& pipeline_receiver = receivers[dst];
      size_t received = 0;

      while (true) {
        auto pending = pipeline_receiver->Receive().get();
        if (pending.Empty())
          break;
        pending.Sync();
        received += pending.GetSegmentSize();
        groute::Segment<data_type> seg(pending.GetSegmentPtr(),
                                       pending.GetSegmentSize());
        pipeline_receiver->ReleaseBuffer(seg, groute::Event());
      }
      total_recv_size += received;
      std::cout << "Dev " << dst << " done, received: "
                << received * sizeof(data_type) / 1024.0f / 1024 << " MB"
                << std::endl;
      //    auto pending = receiver
      //                       ->Receive(groute::Buffer<data_type>(
      //                                     thrust::raw_pointer_cast(recv_data.data()),
      //                                     recv_data.size()),
      //                                 groute::Event())
      //                       .get();
      //    pending.Sync();
    });
  }

  for (auto& th : threads) {
    th.join();
  }

  sw.stop();

  std::cout << "Size: " << size_in_mb << " MB"
            << " Copy time: " << sw.ms() << " Bandwidth: "
            << total_recv_size / 1024.0 / 1024 / (sw.ms() / 1000) << " MB/s"
            << std::endl;
}

struct RankData {
  uint32_t node;
  float rank;

  __host__ __device__ __forceinline__ RankData(uint32_t node, float rank)
      : node(node), rank(rank) {}
  __host__ __device__ __forceinline__ RankData()
      : node(UINT_MAX), rank(-1.0f) {}
  __device__ __host__ __forceinline__ int operator%(int rhs) {
    return node % rhs;
  }
};

template <typename T>
void SplitSegment(groute::Stream& stream, groute::Segment<T>& input_seg,
                  size_t num_split,
                  std::vector<std::shared_ptr<groute::Worklist<T>>>& wl) {
  auto seg_size = input_seg.GetSegmentSize();
  auto* data = input_seg.GetSegmentPtr();

  for (int idx = 0; idx < num_split; idx++) {
    wl[idx]->ResetAsync(stream.hip_stream);

    groute::dev::Worklist<T> d_dev_wl = wl[idx]->DeviceObject();

    LaunchKernel(stream, seg_size, [=] __device__() {
      auto tid = TID_1D;
      auto nthreads = TOTAL_THREADS_1D;

      for (int i = tid; i < seg_size; i += nthreads) {
        if (idx == data[i] % num_split) {
          d_dev_wl.append_warp(data[i]);
        }
      }
    });
  }
}

template <typename T>
__global__ void SplitKernel(hipcub::GridBarrier barrier,
                            groute::Segment<T> input_seg, int num_split,
                            groute::dev::Worklist<T>* output_wl) {
  auto tid = TID_1D;
  auto nthreads = TOTAL_THREADS_1D;
  auto seg_size = input_seg.GetSegmentSize();
  auto* data = input_seg.GetSegmentPtr();

  for (int idx = 0; idx < num_split; idx++) {
    if (tid == 0) {
      output_wl[idx].reset();
      barrier.Sync();
    }
    for (int i = tid; i < seg_size; i += nthreads) {
      if (idx == data[i] % num_split) {
        output_wl[idx].append_warp(data[i]);
      }
    }
    barrier.Sync();
  }
}

template <typename T>
void SplitSegmentFused(groute::Stream& stream,
                       hipcub::GridBarrierLifetime& barrier,
                       groute::Segment<T>& input_seg, size_t num_split,
                       thrust::device_vector<groute::dev::Worklist<T>>& wl) {
  auto seg_size = input_seg.GetSegmentSize();
  auto* data = input_seg.GetSegmentPtr();

  int fused_work_residency = 0;
  int BlockSize = 256;

  hipOccupancyMaxActiveBlocksPerMultiprocessor(&fused_work_residency,
                                                SplitKernel<T>, BlockSize, 0);
  barrier.Setup(fused_work_residency);
  SplitKernel<<<fused_work_residency, BlockSize, 0, stream.hip_stream>>>(
      barrier, input_seg, num_split, thrust::raw_pointer_cast(wl.data()));
}

void TestSplit() {
  groute::Stream stream;
  thrust::device_vector<RankData> ranks(10 * 1000 * 1000);
  int max_split = 4;
  std::vector<std::shared_ptr<groute::Worklist<RankData>>> wls;
  thrust::device_vector<groute::dev::Worklist<RankData>> d_wls;

  std::cout << "Data size: " << ranks.size() * sizeof(RankData) / 1024 / 1024
            << " MB" << std::endl;

  {
    thrust::host_vector<RankData> h_ranks(ranks.size());

    for (int i = 0; i < h_ranks.size(); i++) {
      h_ranks[i].node = i;
    }
    ranks = h_ranks;
  }

  for (int i = 0; i < max_split; i++) {
    auto wl = std::make_shared<groute::Worklist<RankData>>(ranks.size());
    wl->ResetAsync(stream.hip_stream);
    wls.push_back(wl);
    d_wls.push_back(wl->DeviceObject());
  }

  groute::Segment<RankData> input_seg(thrust::raw_pointer_cast(ranks.data()),
                                      ranks.size());
  Stopwatch sw;

  for (int i = 1; i <= max_split; i++) {
    double total_time = 0;
    for (int _ = 0; _ < 100; _++) {
      sw.start();
      SplitSegment(stream, input_seg, i, wls);
      stream.Sync();
      sw.stop();
      total_time += sw.ms();
    }

    std::cout << "split num: " << i << " time: " << total_time / 100
              << std::endl;
  }

  hipcub::GridBarrierLifetime barrier;

  for (int i = 1; i <= max_split; i++) {
    double total_time = 0;
    for (int _ = 0; _ < 100; _++) {
      sw.start();
      SplitSegmentFused(stream, barrier, input_seg, i, d_wls);
      stream.Sync();
      sw.stop();
      total_time += sw.ms();
    }

    std::cout << "split num: " << i << " time: " << total_time / 100
              << std::endl;
  }
}

int main(int argc, char** argv) {
//  gflags::ParseCommandLineFlags(&argc, &argv, true);

//  TestSplit();
//  return 0;
  //  Rings();

  size_t size = 512 * 1024;
  char* src_ptr;
  std::vector<char*> dst_ptrs;
  int src_dev = 0;
  std::vector<int> dst_devs;

  for (int i = 1; i < argc; i++) {
    auto dst = std::string(argv[i]);
    dst_devs.push_back(std::stoi(dst));
  }

  std::vector<int> physical_devs{dst_devs};
  physical_devs.push_back(src_dev);

  for (int physical_dev_i : physical_devs) {
    GROUTE_CUDA_CHECK(hipSetDevice(physical_dev_i));
    for (int physical_dev_j : physical_devs)
      if (physical_dev_i != physical_dev_j)
        hipDeviceEnablePeerAccess(physical_dev_j, 0);
  }

  GROUTE_CUDA_CHECK(hipSetDevice(src_dev));
  GROUTE_CUDA_CHECK(hipMalloc(&src_ptr, size));

  for (int dst_dev : dst_devs) {
    GROUTE_CUDA_CHECK(hipSetDevice(dst_dev));
    char* ptr;
    GROUTE_CUDA_CHECK(hipMalloc(&ptr, size));
    dst_ptrs.push_back(ptr);
  }

  GROUTE_CUDA_CHECK(hipSetDevice(src_dev));

  std::vector<groute::Stream> streams(dst_devs.size());

  {
    double total_time = 0;
    size_t total_size = 0;
    Stopwatch sw;

    for (int iter = 0; iter < 100; iter++) {
      sw.start();
      for (int i = 0; i < dst_devs.size(); i++) {
        GROUTE_CUDA_CHECK(hipMemcpyAsync(dst_ptrs[i], src_ptr, size,
                                          hipMemcpyDeviceToDevice,
                                          streams[i].hip_stream));
      }

      for (auto& stream : streams) {
        stream.Sync();
      }

      sw.stop();
      total_time += sw.ms();
      total_size += size * dst_devs.size();
    }
    std::cout << "Copy with hipMemcpyAsync:" << std::endl;
    std::cout << "Total time: " << total_time << " ms"
              << " Total size: " << (float) total_size / 1024 / 1024 << " MB "
              << " Bandwidth: " << (float) size / 1024 / 1024 / (sw.ms() / 1000)
              << " MB/s" << std::endl;
  }

  std::cout << std::endl;

//  TestSR(dst_devs);
}
